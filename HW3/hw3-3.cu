#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define INF ((1 << 30) - 1)
#define DEV_NO 0
#define B 64  // Block size

int original_n, m;
int *Dist;
hipDeviceProp_t prop;

// Phase 1 Kernel with 2x2 tiling
__global__ void phase1_optimized(int *Dist, int Round, int n) {
    __shared__ int sharedDist[B][B];

    // Each thread handles a 2x2 block
    int tx = threadIdx.x;  // 0 to (B/2 - 1)
    int ty = threadIdx.y;  // 0 to (B/2 - 1)

    int x = tx * 2;
    int y = ty * 2;

    int i_base = Round * B;
    int j_base = Round * B;

    int i = i_base + y;
    int j = j_base + x;

    // Load 2x2 elements into shared memory
    #pragma unroll
    for (int dy = 0; dy < 2; ++dy) {
        #pragma unroll
        for (int dx = 0; dx < 2; ++dx) {
            sharedDist[y + dy][x + dx] = Dist[(i + dy) * n + (j + dx)];
        }
    }
    __syncthreads();

    // Compute Phase 1
    for (int k = 0; k < B; ++k) {
        #pragma unroll
        for (int dy = 0; dy < 2; ++dy) {
            #pragma unroll
            for (int dx = 0; dx < 2; ++dx) {
                int y_idx = y + dy;
                int x_idx = x + dx;
                int new_dist = sharedDist[y_idx][k] + sharedDist[k][x_idx];
                if (new_dist < sharedDist[y_idx][x_idx]) {
                    sharedDist[y_idx][x_idx] = new_dist;
                }
            }
        }
        __syncthreads();
    }

    // Write back to global memory
    #pragma unroll
    for (int dy = 0; dy < 2; ++dy) {
        #pragma unroll
        for (int dx = 0; dx < 2; ++dx) {
            Dist[(i + dy) * n + (j + dx)] = sharedDist[y + dy][x + dx];
        }
    }
}

// Phase 2 Kernel with 2x2 tiling
__global__ void phase2_optimized(int *Dist, int Round, int n) {
    __shared__ int sharedPivot[B][B];
    __shared__ int sharedBlock[B][B];

    int tx = threadIdx.x;  // 0 to (B/2 - 1)
    int ty = threadIdx.y;  // 0 to (B/2 - 1)

    int x = tx * 2;
    int y = ty * 2;

    int i, j;

    if (blockIdx.y == 0) { // Pivot Row
        if (blockIdx.x == Round) return;

        int i_base = Round * B;
        int j_base = blockIdx.x * B;

        i = i_base + y;
        j = j_base + x;

        // Load sharedPivot and sharedBlock
        for (int dy = 0; dy < 2; ++dy) {
            for (int dx = 0; dx < 2; ++dx) {
                sharedPivot[y + dy][x + dx] = Dist[(i + dy) * n + (Round * B + x + dx)];
                sharedBlock[y + dy][x + dx] = Dist[(i + dy) * n + (j + dx)];
            }
        }
    } else { // Pivot Column
        if (blockIdx.x == Round) return;

        int i_base = blockIdx.x * B;
        int j_base = Round * B;

        i = i_base + y;
        j = j_base + x;

        // Load sharedPivot and sharedBlock
        for (int dy = 0; dy < 2; ++dy) {
            for (int dx = 0; dx < 2; ++dx) {
                sharedPivot[y + dy][x + dx] = Dist[(Round * B + y + dy) * n + (j + dx)];
                sharedBlock[y + dy][x + dx] = Dist[(i + dy) * n + (j + dx)];
            }
        }
    }

    __syncthreads();

    // Compute Phase 2
    for (int k = 0; k < B; ++k) {
        for (int dy = 0; dy < 2; ++dy) {
            for (int dx = 0; dx < 2; ++dx) {
                int y_idx = y + dy;
                int x_idx = x + dx;

                int new_dist;
                if (blockIdx.y == 0) {
                    new_dist = sharedPivot[y_idx][k] + sharedBlock[k][x_idx];
                } else {
                    new_dist = sharedBlock[y_idx][k] + sharedPivot[k][x_idx];
                }
                if (new_dist < sharedBlock[y_idx][x_idx]) {
                    sharedBlock[y_idx][x_idx] = new_dist;
                }
            }
        }
        __syncthreads();
    }

    // Write back to global memory
    for (int dy = 0; dy < 2; ++dy) {
        for (int dx = 0; dx < 2; ++dx) {
            Dist[(i + dy) * n + (j + dx)] = sharedBlock[y + dy][x + dx];
        }
    }
}

// Phase 3 Kernel with 2x2 tiling
__global__ void phase3_optimized(int *Dist, int Round, int n, int start_y) {
    if (blockIdx.x == Round || blockIdx.y + start_y == Round) return;

    __shared__ int sharedRow[B][B];
    __shared__ int sharedCol[B][B];

    int tx = threadIdx.x;  // 0 to (B/2 - 1)
    int ty = threadIdx.y;  // 0 to (B/2 - 1)

    int x = tx * 2;
    int y = ty * 2;

    int i_base = (blockIdx.y + start_y) * B;
    int j_base = blockIdx.x * B;

    int i = i_base + y;
    int j = j_base + x;

    // Load sharedRow and sharedCol
    for (int dy = 0; dy < 2; ++dy) {
        for (int dx = 0; dx < 2; ++dx) {
            sharedRow[y + dy][x + dx] = Dist[(i + dy) * n + (Round * B + x + dx)];
            sharedCol[y + dy][x + dx] = Dist[(Round * B + y + dy) * n + (j + dx)];
        }
    }
    __syncthreads();

    // Load current distances
    int current[2][2];
    for (int dy = 0; dy < 2; ++dy) {
        for (int dx = 0; dx < 2; ++dx) {
            current[dy][dx] = Dist[(i + dy) * n + (j + dx)];
        }
    }

    // Compute Phase 3
    for (int k = 0; k < B; ++k) {
        for (int dy = 0; dy < 2; ++dy) {
            for (int dx = 0; dx < 2; ++dx) {
                int y_idx = y + dy;
                int x_idx = x + dx;

                int new_dist = sharedRow[y_idx][k] + sharedCol[k][x_idx];
                if (new_dist < current[dy][dx]) {
                    current[dy][dx] = new_dist;
                }
            }
        }
    }

    // Write back to global memory
    for (int dy = 0; dy < 2; ++dy) {
        for (int dx = 0; dx < 2; ++dx) {
            Dist[(i + dy) * n + (j + dx)] = current[dy][dx];
        }
    }
}

// Input function with padding
void input(char *infile, int *padded_n) {
    FILE *file = fopen(infile, "rb");
    fread(&original_n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    // Calculate padded_n as the smallest multiple of B >= original_n
    *padded_n = ((original_n + B - 1) / B) * B;

    Dist = (int *)malloc((*padded_n) * (*padded_n) * sizeof(int));

    // Initialize the entire matrix to INF
    for (int i = 0; i < (*padded_n) * (*padded_n); ++i)
        Dist[i] = INF;

    // Set Dist[i][i] = 0 for original nodes
    for (int i = 0; i < original_n; ++i)
        Dist[i * (*padded_n) + i] = 0;

    // Read m edges and set the corresponding distances
    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        int a = pair[0], b = pair[1], w = pair[2];
        Dist[a * (*padded_n) + b] = w;
    }
    fclose(file);
}

// Output function writing only original_n x original_n
void output_result(char *outFileName, int padded_n) {
    FILE *outfile = fopen(outFileName, "wb");
    for (int i = 0; i < original_n; ++i)
        fwrite(&Dist[i * padded_n], sizeof(int), original_n, outfile);
    fclose(outfile);
}

int main(int argc, char *argv[]) {
    int padded_n;
    input(argv[1], &padded_n);
    int n = padded_n;  // Update local n to padded_n

    int device_count;
    hipGetDeviceCount(&device_count);  // Get the number of CUDA devices

    int rounds = n / B;

    // Limit the number of devices to the number of rounds
    int active_device_count = device_count;
    if (rounds < device_count) {
        active_device_count = rounds;
    }

    // Allocate array of device pointers for active devices
    int **d_Dist = (int **)malloc(active_device_count * sizeof(int*));

    // Adjust the number of OpenMP threads to the active number of devices
    omp_set_num_threads(active_device_count);

    // Define thread and grid dimensions
    dim3 threads(B / 2, B / 2);  // Each thread handles a 2x2 block
    dim3 grid_phase1(1, 1);
    dim3 grid_phase2(rounds, 2);
    dim3 grid_phase3(rounds, rounds);

    #pragma omp parallel
    {
        int thread_id = omp_get_thread_num();
        hipSetDevice(thread_id);

        // Allocate device memory for each GPU
        hipMalloc((void**)&d_Dist[thread_id], n * n * sizeof(int));

        // Copy the entire Dist matrix to each GPU
        hipMemcpy(d_Dist[thread_id], Dist, n * n * sizeof(int), hipMemcpyHostToDevice);

        for (int r = 0; r < rounds; ++r) {
            // Phase 1 - Only one device needs to perform this
            if (thread_id == 0) {
                phase1_optimized<<<grid_phase1, threads>>>(d_Dist[thread_id], r, n);
                hipDeviceSynchronize();
            }
            #pragma omp barrier  // Ensure phase 1 is completed before phase 2

            // Phase 2
            phase2_optimized<<<grid_phase2, threads>>>(d_Dist[thread_id], r, n);

            // Phase 3
            phase3_optimized<<<grid_phase3, threads>>>(d_Dist[thread_id], r, n, 0);
        }

        // Synchronize before copying data back
        hipDeviceSynchronize();

        // Copy results back to host from device 0
        if (thread_id == 0) {
            hipMemcpy(Dist, d_Dist[thread_id], n * n * sizeof(int), hipMemcpyDeviceToHost);
        }

        // Free device memory
        hipFree(d_Dist[thread_id]);
    }

    // Write the result to the output file
    output_result(argv[2], n);

    // Free host memory
    free(Dist);
    free(d_Dist);

    return EXIT_SUCCESS;
}
